#include "hip/hip_runtime.h"
// Copyright 2016 Illia Olenchenko
#include <iostream>
#include <math.h>

#include "./longStruct.cu"

using namespace std;

// N should be >=3
const int N = 13;
// L should be [4, 10, 52...]
const int L = 4;

__global__ void mykernel (void) {}

int getNextMod(int MOD, int Mp) {
  return (MOD * MOD - 2) % Mp;
}

int main(void) {
  // Calculating probable pure figure
  int Mp = (int)pow(2.0, (double)N) - 1;
  int MOD = L;
  for (int i = 0; i < N - 2; ++i) {
    MOD = getNextMod(MOD, Mp);
    cout << MOD << endl;
  }
  bool isPure = MOD == 0;
  cout << "is pure " << isPure << endl;
  return 0;
}
